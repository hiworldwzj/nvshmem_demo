#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <mpi.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <cstdlib> // 包含 atoi 的头文件


__global__ void simple_shift(int *destination) {
    int mype = nvshmem_my_pe();
    int npes = nvshmem_n_pes();
    int peer = (mype + 1) % npes;

    nvshmem_int_p(destination, mype, peer);
}

int main(int argc, char *argv[]) {
    int num = std::atoi(argv[1]); // 转换为整数
    printf("num: %d\n", num);

    if(num == 0) {
         
        MPI_Init(NULL, NULL);
        char port_name[MPI_MAX_PORT_NAME];
        MPI_Open_port(MPI_INFO_NULL, port_name);
        printf("Port: %s\n", port_name);  // 例如输出 "192.168.1.100:5000"

        // 将端口信息写入文件供Clients读取
        FILE* fp = fopen("port_info.txt", "w");
        fprintf(fp, "%s", port_name);
        fclose(fp);
        MPI_Comm local_self = MPI_COMM_WORLD;
        MPI_Comm client_comm;
        MPI_Comm new_client_comm;
        for (int i = 0; i < 5; i++) {
                MPI_Comm_accept(port_name, MPI_INFO_NULL, 0, local_self, &client_comm);
                printf("Accepted connection from client %d\n", i);
                // 这里可以进行数据交换或其他操作

                printf("wzj1\n");
                MPI_Intercomm_merge(client_comm, 0, &new_client_comm);
                printf("wzj2\n");
                int rank;
                MPI_Comm_rank(new_client_comm, &rank);
                printf("wzj3\n");
                printf("%d rank\n", rank);
                local_self = new_client_comm;


                double local_value, global_sum;
                local_value = 100.0;
                MPI_Allreduce(&local_value, &global_sum, 1, MPI_DOUBLE, MPI_SUM, local_self);
                printf("%f\n", global_sum);
                
        }
    } else {
        char port_name[MPI_MAX_PORT_NAME];
        FILE* fp = fopen("port_info.txt", "r");
        fgets(port_name, MPI_MAX_PORT_NAME, fp);
        fclose(fp);

        MPI_Init(NULL, NULL);
        MPI_Comm local_self = MPI_COMM_WORLD;
        MPI_Comm server_comm;
        MPI_Comm_connect(port_name, MPI_INFO_NULL, 0, local_self, &server_comm);
        int rank;
        MPI_Comm new_server_comm;
        MPI_Intercomm_merge(server_comm, 1, &new_server_comm);
        MPI_Comm_rank(new_server_comm, &rank);
        printf("%d rank\n", rank);
        local_self = new_server_comm;
        MPI_Comm client_comm;
        MPI_Comm new_client_comm;

        double local_value, global_sum;
        local_value = 100.0;
        MPI_Allreduce(&local_value, &global_sum, 1, MPI_DOUBLE, MPI_SUM, local_self);
        printf("%f\n", global_sum);

        for (int i = 0; i < 5; i++) {
            MPI_Comm_accept(port_name, MPI_INFO_NULL, 0, local_self, &client_comm);
            printf("Accepted connection from client %d\n", i);
            // 这里可以进行数据交换或其他操作

            printf("wzj1\n");
            MPI_Intercomm_merge(client_comm, 0, &new_client_comm);
            printf("wzj2\n");
            int rank;
            MPI_Comm_rank(new_client_comm, &rank);
            printf("wzj3\n");
            printf("%d rank", rank);
            local_self = new_client_comm;

            double local_value, global_sum;
            local_value = 100.0;
            MPI_Allreduce(&local_value, &global_sum, 1, MPI_DOUBLE, MPI_SUM, local_self);
            printf("%f\n", global_sum);
        }
    }



    // MPI_Close_port(port_name);
    MPI_Finalize();
    return 0;
}