#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <mpi.h>

__global__ void simple_shift(int *destination) {
    int mype = nvshmem_my_pe();
    int npes = nvshmem_n_pes();
    int peer = (mype + 1) % npes;

    nvshmem_int_p(destination, mype, peer);
}

int main(int argc, char *argv[]) {

    char port_name[MPI_MAX_PORT_NAME];
    FILE* fp = fopen("port_info.txt", "r");
    fgets(port_name, MPI_MAX_PORT_NAME, fp);
    fclose(fp);

    MPI_Init(NULL, NULL);
    MPI_Comm server_comm;
    MPI_Comm_connect(argv[1], MPI_INFO_NULL, 0, MPI_COMM_SELF, &server_comm);
    int rank;
    // MPI_Comm_rank(server_comm, &rank);

    // printf("%d rank", rank);

    // // 接收服务器消息
    // int data;
    // MPI_Recv(&data, 1, MPI_INT, 0, 0, server_comm, MPI_STATUS_IGNORE);
    // printf("Received: %d\n", data);

    MPI_Comm new_server_comm;
    MPI_Intercomm_merge(server_comm, 1, &new_server_comm);

    MPI_Comm_rank(new_server_comm, &rank);

    printf("%d rank\n", rank);

    printf("wzj4");


    MPI_Finalize();
    return 0;
}