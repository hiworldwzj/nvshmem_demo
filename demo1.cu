#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <mpi.h>

__global__ void simple_shift(int *destination) {
    int mype = nvshmem_my_pe();
    int npes = nvshmem_n_pes();
    int peer = (mype + 1) % npes;

    nvshmem_int_p(destination, mype, peer);
}

int main(int argc, char *argv[]) {
    int rank, ndevices;

    nvshmemx_init_attr_t attr;
    MPI_Comm comm = MPI_COMM_WORLD;
    attr.mpi_comm = &comm;

    for(int i =0; i < argc; i++) {
        printf("%s\n", argv[i]);
    }

   extern char **environ; // 声明外部变量 environ
    
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    if(rank == 0) {
        // 遍历并打印所有环境变量
        for (char **env = environ; *env != 0; env++) {
            printf("%s\n", *env);
        };

    }

    int msg;
    hipStream_t stream;

    hipGetDeviceCount(&ndevices);
    hipSetDevice(rank % ndevices);
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
    hipStreamCreate(&stream);

    int *destination = (int *) nvshmem_malloc(sizeof(int));

    simple_shift<<<1, 1, 0, stream>>>(destination);
    nvshmemx_barrier_all_on_stream(stream);
    hipMemcpyAsync(&msg, destination, sizeof(int), hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);
    printf("%d: received message %d\n", nvshmem_my_pe(), msg);

    nvshmem_free(destination);
    nvshmem_finalize();
    MPI_Finalize();
    return 0;
}