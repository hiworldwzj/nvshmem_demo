#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <mpi.h>

__global__ void simple_shift(int *destination) {
    int mype = nvshmem_my_pe();
    int npes = nvshmem_n_pes();
    int peer = (mype + 1) % npes;

    nvshmem_int_p(destination, mype, peer);
}

int main(int argc, char *argv[]) {
    extern char **environ; // 声明外部变量 environ

    int rank, ndevices;
    
    nvshmemx_init_attr_t attr;
    MPI_Comm comm = MPI_COMM_WORLD;
    attr.mpi_comm = &comm;

    // 遍历并打印所有环境变量
    for (char **env = environ; *env != 0; env++) {
        printf("%s\n", *env);
    };
    printf("xxxxxxxxxxxxxxxxxxxxxxxxxxxxx\n");
    
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    // 遍历并打印所有环境变量
    for (char **env = environ; *env != 0; env++) {
        printf("%s\n", *env);
    };

    char port_name[MPI_MAX_PORT_NAME];
    MPI_Open_port(MPI_INFO_NULL, port_name);
    printf("Port: %s\n", port_name);  // 例如输出 "192.168.1.100:5000"


     
    int msg;
    hipStream_t stream;

    hipGetDeviceCount(&ndevices);
    hipSetDevice(rank % ndevices);
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
    hipStreamCreate(&stream);

    int *destination = (int *) nvshmem_malloc(sizeof(int));

    simple_shift<<<1, 1, 0, stream>>>(destination);
    nvshmemx_barrier_all_on_stream(stream);
    hipMemcpyAsync(&msg, destination, sizeof(int), hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);
    printf("%d: received message %d\n", nvshmem_my_pe(), msg);

    nvshmem_free(destination);
    nvshmem_finalize();
    while(true) {

    };
    MPI_Finalize();
    return 0;
}