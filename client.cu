#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <mpi.h>
#include <stdlib.h>
#include <string.h>

__global__ void simple_shift(int *destination) {
    int mype = nvshmem_my_pe();
    int npes = nvshmem_n_pes();
    int peer = (mype + 1) % npes;

    nvshmem_int_p(destination, mype, peer);
}

int main(int argc, char *argv[]) {

    char port_name[MPI_MAX_PORT_NAME];
    FILE* fp = fopen("port_info.txt", "r");
    fgets(port_name, MPI_MAX_PORT_NAME, fp);
    fclose(fp);

    MPI_Init(NULL, NULL);
    MPI_Comm server_comm;
    MPI_Comm_connect(port_name, MPI_INFO_NULL, 0, MPI_COMM_SELF, &server_comm);
    int rank;

    MPI_Comm new_server_comm;
    MPI_Intercomm_merge(server_comm, 1, &new_server_comm);

    MPI_Comm_rank(new_server_comm, &rank);

    printf("%d rank\n", rank);

    double local_value, global_sum;
    local_value = 130.0;
    MPI_Allreduce(&local_value, &global_sum, 1, MPI_DOUBLE, MPI_SUM, new_server_comm);

    printf("%f", global_sum);
    printf("wzj4");


    MPI_Finalize();
    return 0;
}